#include "hip/hip_runtime.h"
#include <Eigen/Dense>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream> 
#include <iomanip> 

void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}


extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

static const char *_cusolverGetErrorEnum(hipsolverStatus_t error)
{
	switch (error)
	{
	case HIPSOLVER_STATUS_SUCCESS:
		return "CUSOLVER_SUCCESS";

	case HIPSOLVER_STATUS_NOT_INITIALIZED:
		return "HIPSOLVER_STATUS_NOT_INITIALIZED";

	case HIPSOLVER_STATUS_ALLOC_FAILED:
		return "HIPSOLVER_STATUS_ALLOC_FAILED";

	case HIPSOLVER_STATUS_INVALID_VALUE:
		return "HIPSOLVER_STATUS_INVALID_VALUE";

	case HIPSOLVER_STATUS_ARCH_MISMATCH:
		return "HIPSOLVER_STATUS_ARCH_MISMATCH";

	case HIPSOLVER_STATUS_EXECUTION_FAILED:
		return "HIPSOLVER_STATUS_EXECUTION_FAILED";

	case HIPSOLVER_STATUS_INTERNAL_ERROR:
		return "HIPSOLVER_STATUS_INTERNAL_ERROR";

	case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	}

	return "<unknown>";
}


inline void __cusolveSafeCall(hipsolverStatus_t err, const char *file, const int line)
{
	if (HIPSOLVER_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSOLVE error in file '%s', line %d, error: %s \nterminating!\n", __FILE__, __LINE__, \
			_cusolverGetErrorEnum(err)); \
			assert(0); \
	}
}

extern "C" void cusolveSafeCall(hipsolverStatus_t err) { __cusolveSafeCall(err, __FILE__, __LINE__); }

namespace LMFAO {
    namespace LinearAlgebra{
        void svdCuda(const Eigen::MatrixXd& a)
        {
           	// --- gesvd only supports Nrows >= Ncols
            // --- column major memory ordering

            int Nrows = a.rows();
            int Ncols = a.cols();

            // --- cuSOLVE input/output parameters/arrays
            int work_size = 0;
            int *devInfo;			gpuErrchk(hipMalloc(&devInfo,	        sizeof(int)));
            
            // --- CUDA solver initialization
            hipsolverHandle_t solver_handle;
            hipsolverDnCreate(&solver_handle);

            // --- Setting the host, Nrows x Ncols matrix
            double *h_A = (double *)malloc(Nrows * Ncols * sizeof(double));
            for(int j = 0; j < Nrows; j++)
                for(int i = 0; i < Ncols; i++)
                {
                    //h_A[j + i*Nrows] = (i + j*j) * sqrt((double)(i + j));
                    h_A[j + i*Nrows] = a(i, j);
                }
            // --- Setting the device matrix and moving the host matrix to the device
            double *d_A;			gpuErrchk(hipMalloc(&d_A,		Nrows * Ncols * sizeof(double)));
            gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));

            // --- host side SVD results space
            double *h_U = (double *)malloc(Nrows * Nrows     * sizeof(double));
            double *h_V = (double *)malloc(Ncols * Ncols     * sizeof(double));
            double *h_S = (double *)malloc(min(Nrows, Ncols) * sizeof(double));

            // --- device side SVD workspace and matrices
            double *d_U;			gpuErrchk(hipMalloc(&d_U,	Nrows * Nrows     * sizeof(double)));
            double *d_V;			gpuErrchk(hipMalloc(&d_V,	Ncols * Ncols	  * sizeof(double)));
            double *d_S;			gpuErrchk(hipMalloc(&d_S,	min(Nrows, Ncols) * sizeof(double)));
            
            std::cout << "Init" << std::endl;
            // --- CUDA SVD initialization
            cusolveSafeCall(hipsolverDnDgesvd_bufferSize(solver_handle, Nrows, Ncols, &work_size));
            double *work;	gpuErrchk(hipMalloc(&work, work_size * sizeof(double)));
            
            std::cout << "Exec" << std::endl;
            // --- CUDA SVD execution
            cusolveSafeCall(hipsolverDnDgesvd(solver_handle, 'A', 'A', Nrows, Ncols, d_A, Nrows, d_S, d_U, Nrows, d_V, Ncols, work, work_size, NULL, devInfo));
            int devInfo_h = 0;	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
            if (devInfo_h != 0) std::cout	<< "Unsuccessful SVD execution\n\n";
            std::cout << "Copy" << std::endl;
            // --- Moving the results from device to host
            gpuErrchk(hipMemcpy(h_S, d_S, min(Nrows, Ncols) * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(h_U, d_U, Nrows * Nrows     * sizeof(double), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(h_V, d_V, Ncols * Ncols     * sizeof(double), hipMemcpyDeviceToHost));

            std::cout << "Singular values\n";
            for(int i = 0; i < min(Nrows, Ncols); i++) 
                std::cout << "d_S["<<i<<"] = " << std::setprecision(15) << h_S[i] << std::endl;

            std::cout << "\nLeft singular vectors - For y = A * x, the columns of U span the space of y\n";
            for(int j = 0; j < Nrows; j++) {
                printf("\n");
                for(int i = 0; i < Nrows; i++)
                    printf("U[%i,%i]=%f\n",i,j,h_U[j*Nrows + i]);
            }

            std::cout << "\nRight singular vectors - For y = A * x, the columns of V span the space of x\n";
            for(int i = 0; i < Ncols; i++) {
                printf("\n");
                for(int j = 0; j < Ncols; j++)
                    printf("V[%i,%i]=%f\n",i,j,h_V[j*Ncols + i]);
            }

            hipsolverDnDestroy(solver_handle);
        }
    }
}

